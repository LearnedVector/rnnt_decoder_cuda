#include<jointnet.hpp>
#include<hparams.hpp>
#include<data_types.hpp>

#include<iostream>
#include<vector>
#include<string>
#include<logger.hpp>
// #include<utils.hpp>
#include<hipblas.h>


using namespace s2t::decodernet;
using namespace s2t::common;
using namespace std;

jointnet::jointnet()
{}

void jointnet::init(hipdnnHandle_t& cudnn, const std::string& base_model_path)
{
	size_t dense_1_hidden_size = 0;
	size_t dense_2_hidden_size = 0;

	// initialize dense_1
	{
		auto kernel_weight = cnpy::npy_load(base_model_path + hparams::joint_net_dense_0_kernel); 
		auto bias_weight = cnpy::npy_load(base_model_path + hparams::joint_net_dense_0_bias);

        dense_1.init(kernel_weight, bias_weight);
        
		dense_1_hidden_size = kernel_weight.shape[1]; 
		
		// cout << "dense_1_hidden_size: " << dense_1_hidden_size << endl;
	}
	
	// initialise relu activation layer
	{
		activation_t.init(1, dense_1_hidden_size, 1, 1, HIPDNN_ACTIVATION_RELU);
	}
    
	// initialize dense_2
	{
		auto kernel_weight = cnpy::npy_load(base_model_path + hparams::joint_net_dense_1_kernel); 
		auto bias_weight = cnpy::npy_load(base_model_path + hparams::joint_net_dense_1_bias);

		dense_2.init(kernel_weight, bias_weight);

		dense_2_hidden_size = kernel_weight.shape[1]; 
	}

	// intitlaize gpu variables
	{
		var1.init(hparams::max_input_size, dense_1_hidden_size);
		// var2.init(hparams::max_input_size, dense_2_hidden_size);
		hipdnnCreateTensorDescriptor(&tExamples);
		hipdnnSetTensor4dDescriptor(tExamples, hipdnnTensorFormat_t::HIPDNN_TENSOR_NCHW, hipdnnDataType_t::HIPDNN_DATA_FLOAT, 1, dense_2_hidden_size, 1, 1);
	}	
}


void jointnet::operator() (hipdnnHandle_t& cudnn, gpu_float_array& input, gpu_float_array& output)
{
    // reset and reshape the Vars based on input size
	var1.reset();
	var1.reshape(1, var1.shape[1]);

	dense_1(cudnn, input, var1);
	activation_t(cudnn, var1);
	dense_2(cudnn, var1, output);
	hipdnnStatus_t status = hipdnnSoftmaxForward(cudnn, hipdnnSoftmaxAlgorithm_t::HIPDNN_SOFTMAX_LOG, hipdnnSoftmaxMode_t::HIPDNN_SOFTMAX_MODE_INSTANCE, &one, tExamples, output.ptr, &zero, tExamples, output.ptr);
}

jointnet::~jointnet()
{}