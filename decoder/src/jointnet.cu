#include "hip/hip_runtime.h"
#include<jointnet.hpp>
#include<hparams.hpp>
#include<data_types.hpp>

#include<iostream>
#include<vector>
#include<string>
#include<logger.hpp>
// #include<utils.hpp>
#include<hipblas.h>


using namespace s2t::decodernet;
using namespace s2t::common;
using namespace std;

jointnet::jointnet()
{}

void jointnet::init(hipdnnHandle_t& cudnn, const std::string& base_model_path)
{
	size_t dense_1_hidden_size = 0;
	size_t dense_2_hidden_size = 0;

	// initialize dense_1
	{
		auto kernel_weight = cnpy::npy_load(base_model_path + hparams::joint_net_dense_0_kernel); 
		auto bias_weight = cnpy::npy_load(base_model_path + hparams::joint_net_dense_0_bias);

        dense_1.init(kernel_weight, bias_weight);
        
		dense_1_hidden_size = kernel_weight.shape[1]; 
		
		// cout << "dense_1_hidden_size: " << dense_1_hidden_size << endl;
	}
	
	// initialise relu activation layer
	{
		activation_t.init(1, dense_1_hidden_size, 1, 1, HIPDNN_ACTIVATION_RELU);
	}
    
	// initialize dense_2
	{
		auto kernel_weight = cnpy::npy_load(base_model_path + hparams::joint_net_dense_1_kernel); 
		auto bias_weight = cnpy::npy_load(base_model_path + hparams::joint_net_dense_1_bias);

		dense_2.init(kernel_weight, bias_weight);

		dense_2_hidden_size = kernel_weight.shape[1]; 
	}

	// intitlaize gpu variables
	{
		var1.init(hparams::max_input_size, dense_1_hidden_size);
		// var2.init(hparams::max_input_size, dense_2_hidden_size);
		hipdnnCreateTensorDescriptor(&tExamples);
		hipdnnSetTensor4dDescriptor(tExamples, hipdnnTensorFormat_t::HIPDNN_TENSOR_NCHW, hipdnnDataType_t::HIPDNN_DATA_FLOAT, 1, dense_2_hidden_size, 1, 1);
	}	
}


void jointnet::operator() (hipdnnHandle_t& cudnn, gpu_float_array& input, gpu_float_array& output)
{
    // reset and reshape the Vars based on input size
	var1.reset();
	var1.reshape(1, var1.shape[1]);
	// var2.reset();
	// var2.reshape(1, var2.shape[1]);

	dense_1(cudnn, input, var1);
	// log_e("dense 1 output", var1.log("cpp_joint_net_dense_1.npy.npy"));
	activation_t(cudnn, var1);
	// log_e("dense 1 relu output", var1.log("cpp_joint_net_dense_1_relu.npy.npy"));

	dense_2(cudnn, var1, output);
	// log_e("dense 2 output", output.log("cpp_joint_net_dense_2.npy"));

	// cout << output.shape[1] << endl;

	hipdnnStatus_t status = hipdnnSoftmaxForward(cudnn, hipdnnSoftmaxAlgorithm_t::HIPDNN_SOFTMAX_LOG, hipdnnSoftmaxMode_t::HIPDNN_SOFTMAX_MODE_INSTANCE, &one, tExamples, output.ptr, &zero, tExamples, output.ptr);
	
	// cout << HIPDNN_STATUS_BAD_PARAM << " " << HIPDNN_STATUS_SUCCESS << " : " << status << endl;
	// log_e("dense 2 softmax output", output.log("cpp_joint_net_dense_2_softmax.npy.npy"));
}

jointnet::~jointnet()
{}